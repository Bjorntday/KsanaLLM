#include "hip/hip_runtime.h"
/*
 * Copyright 2024 Tencent Inc.  All rights reserved.
 */

#include <random>
#include <sstream>

#include <gtest/gtest.h>

#include "3rdparty/half/include/half.hpp"
#include "cuda_fp8_utils.h"
#include "tests/kernels/nvidia/utils/testsuit_base.h"
namespace llm_kernels {
namespace nvidia {
namespace test {
class LLMKernelsNvidiaUtilsTestSuit : public NvidiaTestSuitBase {
 public:
  void SetUp() override { NvidiaTestSuitBase::SetUp(); }

  void TearDown() override { NvidiaTestSuitBase::TearDown(); }

 protected:
  using NvidiaTestSuitBase::stream;
};

#ifdef ENABLE_FP8
TEST_F(LLMKernelsNvidiaUtilsTestSuit, ComputeFP8QuantizeScaleTest) {
  // <num_channels, channel_size>
  using testcase_t = std::pair<size_t, size_t>;
  std::vector<testcase_t> testcases = {{1, 31}, {1, 16383}, {16383, 7}, {7, 16383}, {16383, 1}, {31, 1}};
  for (testcase_t& shape : testcases) {
    int32_t num_channels = shape.first;
    int32_t channel_size = shape.second;

    BufferMeta input = CreateBuffer<half>(MemoryType::MEMORY_GPU, {shape.first, shape.second}, /*is_random_init*/ true);
    half* input_ptr = reinterpret_cast<half*>(input.data_ptr);

    BufferMeta input_host = CopyToHost<half>(input);
    half* input_host_ptr = reinterpret_cast<half*>(input_host.data_ptr);

    BufferMeta output = CreateBuffer<float>(MemoryType::MEMORY_GPU, {shape.first}, /*is_random_init*/ false);
    float* output_ptr = reinterpret_cast<float*>(output.data_ptr);
    InvokeComputeFP8QuantizeScale(output_ptr, input_ptr, num_channels, channel_size, stream);

    BufferMeta output_host = CopyToHost<float>(output);
    float* output_host_ptr = static_cast<float*>(output_host.data_ptr);

    CHECK_NVIDIA_CUDA_ERROR(hipStreamSynchronize(stream));
    for (int n = 0; n < num_channels; ++n) {
      float channel_max = 0.f;
      for (int k = 0; k < channel_size; ++k) {
        float val = fabs(static_cast<float>(input_host_ptr[n * channel_size + k]));
        channel_max = std::max(val, channel_max);
      }
      channel_max = std::max(channel_max / FP8_E4M3_MAX, FP8_E4M3_MIN_SCALE);
      EXPECT_TRUE(AlmostEqual(channel_max, output_host_ptr[n], 1e-6));
    }
  }
}

TEST_F(LLMKernelsNvidiaUtilsTestSuit, QuantizeMatrixTest) {
  // <num_channels, channel_size>
  using testcase_t = std::pair<size_t, size_t>;
  std::vector<testcase_t> testcases = {{1, 31}, {1, 16383}, {16383, 7}, {7, 16383}, {16383, 1}, {31, 1}};
  for (testcase_t& shape : testcases) {
    int32_t num_channels = shape.first;
    int32_t channel_size = shape.second;

    BufferMeta input = CreateBuffer<half>(MemoryType::MEMORY_GPU, {shape.first, shape.second}, true);
    half* input_ptr = reinterpret_cast<half*>(input.data_ptr);

    BufferMeta input_host = CopyToHost<half>(input);
    half* input_host_ptr = reinterpret_cast<half*>(input_host.data_ptr);

    BufferMeta scale = CreateBuffer<float>(MemoryType::MEMORY_GPU, {shape.first}, true, FP8_E4M3_MIN_SCALE, 1.f);
    float* scale_ptr = reinterpret_cast<float*>(scale.data_ptr);

    BufferMeta scale_host = CopyToHost<float>(scale);
    float* scale_host_ptr = reinterpret_cast<float*>(scale_host.data_ptr);

    BufferMeta output =
        CreateBuffer<__hip_fp8_e4m3_fnuz>(MemoryType::MEMORY_GPU, {shape.first, shape.second}, /*is_random_init*/ false);
    __hip_fp8_e4m3_fnuz* output_ptr = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(output.data_ptr);

    InvokeQuantizeMatrix(output_ptr, scale_ptr, input_ptr, num_channels, channel_size, stream);

    BufferMeta output_host = CopyToHost<__hip_fp8_e4m3_fnuz>(output);

    __hip_fp8_e4m3_fnuz* output_host_ptr = static_cast<__hip_fp8_e4m3_fnuz*>(output_host.data_ptr);

    CHECK_NVIDIA_CUDA_ERROR(hipStreamSynchronize(stream));

    for (int n = 0; n < num_channels; ++n) {
      for (int k = 0; k < channel_size; ++k) {
        float val = static_cast<float>(input_host_ptr[n * channel_size + k]);
        val = std::min(std::max(val / scale_host_ptr[n], -FP8_E4M3_MAX), FP8_E4M3_MAX);
        val = static_cast<float>(static_cast<__hip_fp8_e4m3_fnuz>(val));
        EXPECT_TRUE(AlmostEqual(val, static_cast<float>(output_host_ptr[n * channel_size + k]), 1e-3, 1e-4));
      }
    }
  }
}
#endif  // ENABLE_FP8

}  // namespace test
}  // namespace nvidia
}  // namespace llm_kernels
