#include "hip/hip_runtime.h"
/*
 * Copyright 2024 Tencent Inc.  All rights reserved.
 */

#include <random>
#include <sstream>

#include <gtest/gtest.h>

#include "3rdparty/half/include/half.hpp"
#include "cuda_fp8_utils.h"
#include "tests/kernels/nvidia/utils/testsuit_base.h"
namespace llm_kernels {
namespace nvidia {
namespace test {
class LLMKernelsNvidiaUtilsTestSuit : public NvidiaTestSuitBase {
 public:
  void SetUp() override { NvidiaTestSuitBase::SetUp(); }

  void TearDown() override { NvidiaTestSuitBase::TearDown(); }

 protected:
  using NvidiaTestSuitBase::stream;
};

#ifdef ENABLE_FP8
TEST_F(LLMKernelsNvidiaUtilsTestSuit, ComputeFP8QuantizeScaleTest) {
  // <num_channels, channel_size>
  using testcase_t = std::pair<size_t, size_t>;
  std::vector<testcase_t> testcases = {{1, 31}, {1, 16383}, {16383, 7}, {7, 16383}, {16383, 1}, {31, 1}};
  for (testcase_t& shape : testcases) {
    int32_t num_channels = shape.first;
    int32_t channel_size = shape.second;

    BufferMeta input = CreateBuffer<half>(MemoryType::MEMORY_GPU, {shape.first, shape.second}, /*is_random_init*/ true);
    half* input_ptr = reinterpret_cast<half*>(input.data_ptr);

    BufferMeta input_host = CopyToHost<half>(input);
    half* input_host_ptr = reinterpret_cast<half*>(input_host.data_ptr);

    BufferMeta output = CreateBuffer<float>(MemoryType::MEMORY_GPU, {shape.first}, /*is_random_init*/ false);
    float* output_ptr = reinterpret_cast<float*>(output.data_ptr);
    InvokeComputeFP8QuantizeScale(output_ptr, input_ptr, num_channels, channel_size, stream);

    BufferMeta output_host = CopyToHost<float>(output);
    float* output_host_ptr = static_cast<float*>(output_host.data_ptr);

    CHECK_NVIDIA_CUDA_ERROR(hipStreamSynchronize(stream));
    for (int n = 0; n < num_channels; ++n) {
      float channel_max = 0.f;
      for (int k = 0; k < channel_size; ++k) {
        float val = fabs(static_cast<float>(input_host_ptr[n * channel_size + k]));
        channel_max = std::max(val, channel_max);
      }
      channel_max = std::max(channel_max / FP8_E4M3_MAX, FP8_E4M3_MIN_SCALE);
      EXPECT_TRUE(AlmostEqual(channel_max, output_host_ptr[n], 1e-6));
    }
  }
}

TEST_F(LLMKernelsNvidiaUtilsTestSuit, QuantizeMatrixTest) {
  // <num_channels, channel_size>
  using testcase_t = std::pair<size_t, size_t>;
  std::vector<testcase_t> testcases = {{1, 31}, {1, 16383}, {16383, 7}, {7, 16383}, {16383, 1}, {31, 1}};
  for (testcase_t& shape : testcases) {
    int32_t num_channels = shape.first;
    int32_t channel_size = shape.second;

    BufferMeta input = CreateBuffer<half>(MemoryType::MEMORY_GPU, {shape.first, shape.second}, true);
    half* input_ptr = reinterpret_cast<half*>(input.data_ptr);

    BufferMeta input_host = CopyToHost<half>(input);
    half* input_host_ptr = reinterpret_cast<half*>(input_host.data_ptr);

    BufferMeta scale = CreateBuffer<float>(MemoryType::MEMORY_GPU, {shape.first}, true, FP8_E4M3_MIN_SCALE, 1.f);
    float* scale_ptr = reinterpret_cast<float*>(scale.data_ptr);

    BufferMeta scale_host = CopyToHost<float>(scale);
    float* scale_host_ptr = reinterpret_cast<float*>(scale_host.data_ptr);

    BufferMeta output =
        CreateBuffer<__hip_fp8_e4m3_fnuz>(MemoryType::MEMORY_GPU, {shape.first, shape.second}, /*is_random_init*/ false);
    __hip_fp8_e4m3_fnuz* output_ptr = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(output.data_ptr);

    InvokeQuantizeMatrix(output_ptr, scale_ptr, input_ptr, num_channels, channel_size, stream);

    BufferMeta output_host = CopyToHost<__hip_fp8_e4m3_fnuz>(output);

    __hip_fp8_e4m3_fnuz* output_host_ptr = static_cast<__hip_fp8_e4m3_fnuz*>(output_host.data_ptr);

    CHECK_NVIDIA_CUDA_ERROR(hipStreamSynchronize(stream));

    for (int n = 0; n < num_channels; ++n) {
      for (int k = 0; k < channel_size; ++k) {
        float val = static_cast<float>(input_host_ptr[n * channel_size + k]);
        val = std::min(std::max(val / scale_host_ptr[n], -FP8_E4M3_MAX), FP8_E4M3_MAX);
        val = static_cast<float>(static_cast<__hip_fp8_e4m3_fnuz>(val));
        EXPECT_TRUE(AlmostEqual(val, static_cast<float>(output_host_ptr[n * channel_size + k]), 1e-3, 1e-4));
      }
    }
  }
}

TEST_F(LLMKernelsNvidiaUtilsTestSuit, ReScaleFp8E4m3Test) {
  // <num_channels, channel_size>
  std::vector<size_t> testcases = {1, 7, 31, 16383};
  for (size_t num_elems : testcases) {
    BufferMeta input = CreateBuffer<__hip_fp8_e4m3_fnuz>(MemoryType::MEMORY_GPU, {num_elems}, true);
    __hip_fp8_e4m3_fnuz* input_ptr = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(input.data_ptr);

    BufferMeta input_host = CopyToHost<__hip_fp8_e4m3_fnuz>(input);
    __hip_fp8_e4m3_fnuz* input_host_ptr = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(input_host.data_ptr);

    BufferMeta input_scale = CreateBuffer<float>(MemoryType::MEMORY_GPU, {1}, true, FP8_E4M3_MIN_SCALE, 1.f);
    float* input_scale_ptr = reinterpret_cast<float*>(input_scale.data_ptr);

    BufferMeta input_scale_host = CopyToHost<float>(input_scale);
    float* input_scale_host_ptr = reinterpret_cast<float*>(input_scale_host.data_ptr);

    BufferMeta output_scale = CreateBuffer<float>(MemoryType::MEMORY_GPU, {1}, true, FP8_E4M3_MIN_SCALE, 1.f);
    float* output_scale_ptr = reinterpret_cast<float*>(output_scale.data_ptr);

    BufferMeta output_scale_host = CopyToHost<float>(output_scale);
    float* output_scale_host_ptr = reinterpret_cast<float*>(output_scale_host.data_ptr);

    BufferMeta output = CreateBuffer<__hip_fp8_e4m3_fnuz>(MemoryType::MEMORY_GPU, {num_elems}, /*is_random_init*/ false);
    __hip_fp8_e4m3_fnuz* output_ptr = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(output.data_ptr);

    InvokeRescaleFp8E4m3(input_ptr, output_ptr, (int)num_elems, input_scale_ptr, output_scale_ptr, stream);

    BufferMeta output_host = CopyToHost<__hip_fp8_e4m3_fnuz>(output);
    __hip_fp8_e4m3_fnuz* output_host_ptr = static_cast<__hip_fp8_e4m3_fnuz*>(output_host.data_ptr);

    CHECK_NVIDIA_CUDA_ERROR(hipStreamSynchronize(stream));

    for (size_t n = 0; n < num_elems; ++n) {
      float val = static_cast<float>(input_host_ptr[n]);
      val = std::min(std::max(val * input_scale_host_ptr[0] / output_scale_host_ptr[0], -FP8_E4M3_MAX), FP8_E4M3_MAX);
      val = static_cast<float>(static_cast<__hip_fp8_e4m3_fnuz>(val));
      EXPECT_TRUE(AlmostEqual(val, static_cast<float>(output_host_ptr[n]), 1e-3, 1e-4));
    }
  }
}
#endif  // ENABLE_FP8

}  // namespace test
}  // namespace nvidia
}  // namespace llm_kernels
