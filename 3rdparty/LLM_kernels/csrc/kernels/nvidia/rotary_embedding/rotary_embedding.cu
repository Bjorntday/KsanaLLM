#include "hip/hip_runtime.h"
/*
 * Modify from
 * https://github.com/vllm-project/vllm/blob/v0.2.3/csrc/pos_encoding_kernels.cu
 * Copyright (c) 2024, Tencent Inc.
 * Copyright (c) 2023, The vLLM team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "rotary_embedding.h"

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <cmath>

#include "csrc/utils/nvidia/cuda_utils.h"

using namespace llm_kernels::utils;

namespace llm_kernels {
namespace nvidia {

template <typename T, bool IS_NEOX>
inline __device__ void ApplyRotaryEmbedding(T* __restrict__ arr, const T* __restrict__ cos_ptr,
                                            const T* __restrict__ sin_ptr, int rot_offset, int embed_dim) {
  int x_index, y_index;
  T cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = __ldg(cos_ptr + x_index);
    sin = __ldg(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = __ldg(cos_ptr + x_index / 2);
    sin = __ldg(sin_ptr + x_index / 2);
  }

  const T x = arr[x_index];
  const T y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template <typename T, bool IS_NEOX>
__global__ void InvokeRotaryEmbeddingKernel(
    const int64_t* __restrict__ positions,  // [batch_size, seq_len] or [num_tokens]
    const int64_t* __restrict__ mask,       // [batch_size, seq_len] or [num_tokens]
    T* __restrict__ query,  // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
    T* __restrict__ key,    // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads,
                            // head_size]
    const T* __restrict__ cos_sin_cache,  // [max_position_embeddings, 2, rotary_dim // 2]
    const int rotary_dim, const int64_t query_stride, const int64_t key_stride, const int num_heads,
    const int num_kv_heads, const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  int64_t mask_i = mask[token_idx];
  if (mask_i == 0) {
    return;
  }
  const T* cache_ptr = cos_sin_cache + pos * rotary_dim;

  const int embed_dim = rotary_dim / 2;
  const T* cos_ptr = cache_ptr;
  const T* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    ApplyRotaryEmbedding<T, IS_NEOX>(query + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    ApplyRotaryEmbedding<T, IS_NEOX>(key + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
  }
}

template <typename T>
void LaunchRotaryEmbedding(const RotaryEmbeddingParam<T>& params) {
  dim3 grid(params.num_tokens_);
  dim3 block(std::min(params.num_heads * params.rotary_dim / 2, 512));
  if (params.is_neox) {
    InvokeRotaryEmbeddingKernel<T, true><<<grid, block, 0, params.stream>>>(
        params.positions, params.mask, params.query_, params.key_, params.cos_sin_cache, params.rotary_dim,
        params.query_stride, params.key_stride, params.num_heads, params.num_kv_heads, params.head_size);
  } else {
    InvokeRotaryEmbeddingKernel<T, false><<<grid, block, 0, params.stream>>>(
        params.positions, params.mask, params.query_, params.key_, params.cos_sin_cache, params.rotary_dim,
        params.query_stride, params.key_stride, params.num_heads, params.num_kv_heads, params.head_size);
  }
}

template void LaunchRotaryEmbedding<float>(const RotaryEmbeddingParam<float>& params);
template void LaunchRotaryEmbedding<half>(const RotaryEmbeddingParam<half>& params);
template void LaunchRotaryEmbedding<__hip_bfloat16>(const RotaryEmbeddingParam<__hip_bfloat16>& params);

template <typename T>
__global__ void InvokeComputeCosSinWithCacheKernel(T* __restrict__ cos_sin_cache, const int rotary_dim,
                                                   const int max_position_embeddings, const float base,
                                                   const float scaling) {
  int pos = blockIdx.x;
  for (int rid = threadIdx.x; rid < rotary_dim / 2; rid += blockDim.x) {
    float inv_freq = 1.0 / pow(base, rid * 2 / (float)rotary_dim);
    float freq = pos * inv_freq / scaling;
    cos_sin_cache[pos * rotary_dim + rid] = (T)cos(freq);
    cos_sin_cache[pos * rotary_dim + rotary_dim / 2 + rid] = (T)sin(freq);
  }
}

template <typename T>
__global__ void InvokeComputeMultiFreqCosSinWithCacheKernel(T* __restrict__ cos_sin_cache, const int rotary_dim,
                                                         const int max_position_embeddings, const float base,
                                                         const float scaling, const float low_freq_factor,
                                                         const float high_freq_factor,
                                                         const int original_max_position_embeddings) {
  int pos = blockIdx.x;
  float low_freq_wavelen = (float)original_max_position_embeddings / low_freq_factor;
  float high_freq_wavelen = (float)original_max_position_embeddings / high_freq_factor;
  for (int rid = threadIdx.x; rid < rotary_dim / 2; rid += blockDim.x) {
    float inv_freq = 1.0f / pow(base, rid * 2 / (float)rotary_dim);
    float wavelen = 2.0f * M_PI / inv_freq;
    float freq = inv_freq;
    // Same logic as :
    // https://github.com/vllm-project/vllm/blob/c5df56f88bc8a5a32a0534793f48182a333aeca4/vllm/model_executor/layers/rotary_embedding.py#L742
    if (wavelen < high_freq_wavelen) {
      freq = inv_freq;
    } else if (wavelen > low_freq_wavelen) {
      freq = inv_freq / scaling;
    } else {
      float smooth =
          ((float)original_max_position_embeddings / wavelen - low_freq_factor) / (high_freq_factor - low_freq_factor);
      freq = (1.0f - smooth) * inv_freq / scaling + smooth * inv_freq;
    }
    freq = pos * freq;
    cos_sin_cache[pos * rotary_dim + rid] = (T)cos(freq);
    cos_sin_cache[pos * rotary_dim + rotary_dim / 2 + rid] = (T)sin(freq);
  }
}

template <typename T>
void ComputeCosSinWithCache(const RotaryEmbeddingParam<T>& params) {
  size_t extend_max_len = params.max_position_embeddings;
  dim3 block(std::min(params.rotary_dim / 2, DEFAULT_CUDA_BLOCK_THREADS_NUM));

  float base = params.base;
  float scaling = 1.0f;
  // Same logic as :
  // https://github.com/vllm-project/vllm/blob/523e30ea0c5abcb447763dcd9a77b54d5c5f3239/vllm/model_executor/layers/rotary_embedding.py#L219
  if (params.rotary_embedding_type == RotaryEmbeddingType::DYNAMIC_NTK_SCALING) {
    extend_max_len = params.max_position_embeddings * params.scaling_factor;
    base = std::pow(params.base * ((params.scaling_factor * extend_max_len / params.max_position_embeddings) -
                                   (params.scaling_factor - 1)),
                    (params.rotary_dim / (params.rotary_dim - 2)));
  }
  if (params.rotary_embedding_type == RotaryEmbeddingType::LINEAR_SCALING) {
    extend_max_len = params.max_position_embeddings * params.scaling_factor;
    scaling = params.scaling_factor;
  }
  if (params.rotary_embedding_type == RotaryEmbeddingType::MULTIFREQ_SCALING) {
    scaling = params.scaling_factor;
    float low_freq_factor = params.low_freq_factor;
    float high_freq_factor = params.high_freq_factor;
    int original_max_position_embeddings = params.original_max_position_embeddings;

    dim3 grid(extend_max_len);
    InvokeComputeMultiFreqCosSinWithCacheKernel<T>
        <<<grid, block, 0, params.stream>>>(params.cos_sin_cache, params.rotary_dim, extend_max_len, base, scaling,
                                            low_freq_factor, high_freq_factor, original_max_position_embeddings);
  } else {
    dim3 grid(extend_max_len);
    InvokeComputeCosSinWithCacheKernel<T>
        <<<grid, block, 0, params.stream>>>(params.cos_sin_cache, params.rotary_dim, extend_max_len, base, scaling);
  }
}

template void ComputeCosSinWithCache<float>(const RotaryEmbeddingParam<float>& params);
template void ComputeCosSinWithCache<half>(const RotaryEmbeddingParam<half>& params);
template void ComputeCosSinWithCache<__hip_bfloat16>(const RotaryEmbeddingParam<__hip_bfloat16>& params);

template <typename T>
void RotaryEmbeddingCuda<T>::SetInput(
    const int64_t* positions,  // [batch_size, seq_len] or [num_tokens]
    const int64_t* mask,
    T* query,  // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
    T* key,    // [batch_size, seq_len, num_kv_heads * head_size] or [num_tokens, num_kv_heads * head_size]
    int num_tokens, hipStream_t& stream) {
  params_.positions = positions;
  params_.mask = mask;
  params_.query_ = query;
  params_.key_ = key;
  params_.num_tokens_ = num_tokens;
  params_.stream = stream;
}

template void RotaryEmbeddingCuda<float>::SetInput(const int64_t* positions, const int64_t* mask, float* query,
                                                   float* key, int num_tokens, hipStream_t& stream);
template void RotaryEmbeddingCuda<half>::SetInput(const int64_t* positions, const int64_t* mask, half* query, half* key,
                                                  int num_tokens, hipStream_t& stream);
template void RotaryEmbeddingCuda<__hip_bfloat16>::SetInput(const int64_t* positions, const int64_t* mask,
                                                           __hip_bfloat16* query, __hip_bfloat16* key, int num_tokens,
                                                           hipStream_t& stream);

template <typename T>
void RotaryEmbeddingCuda<T>::Forward() {
  LaunchRotaryEmbedding(params_);
}

template void RotaryEmbeddingCuda<float>::Forward();
template void RotaryEmbeddingCuda<half>::Forward();
template void RotaryEmbeddingCuda<__hip_bfloat16>::Forward();

template <typename T>
void RotaryEmbeddingCuda<T>::SetConfig(T* cos_sin_cache, const int rotary_dim, const int max_position_embeddings,
                                       const float base, const int head_size, const int num_heads,
                                       const int num_kv_heads, const int stride_size, const bool is_neox,
                                       hipStream_t& stream, const RotaryEmbeddingType rotary_embedding_type,
                                       const float scaling_factor, const float low_freq_factor,
                                       const float high_freq_factor, const int original_max_position_embeddings) {
  params_.cos_sin_cache = cos_sin_cache;
  params_.rotary_dim = rotary_dim;
  params_.max_position_embeddings = max_position_embeddings;
  params_.base = base;
  params_.head_size = head_size;
  params_.num_heads = num_heads;
  params_.num_kv_heads = num_kv_heads;
  params_.is_neox = is_neox;
  params_.stream = stream;
  params_.query_stride = stride_size;
  params_.key_stride = stride_size;
  params_.rotary_embedding_type = rotary_embedding_type;
  params_.scaling_factor = scaling_factor;
  params_.low_freq_factor = low_freq_factor;
  params_.high_freq_factor = high_freq_factor;
  params_.original_max_position_embeddings = original_max_position_embeddings;
  ComputeCosSinWithCache(params_);
}

template void RotaryEmbeddingCuda<float>::SetConfig(float* cos_sin_cache, const int rotary_dim,
                                                    const int max_position_embeddings, const float base,
                                                    const int head_size, const int num_heads, const int num_kv_heads,
                                                    const int stride_size, const bool is_neox, hipStream_t& stream,
                                                    const RotaryEmbeddingType rotary_embedding_type,
                                                    const float scaling_factor, const float low_freq_factor,
                                                    const float high_freq_factor, const int original_max_position_embeddings);
template void RotaryEmbeddingCuda<half>::SetConfig(half* cos_sin_cache, const int rotary_dim,
                                                   const int max_position_embeddings, const float base,
                                                   const int head_size, const int num_heads, const int num_kv_heads,
                                                   const int stride_size, const bool is_neox, hipStream_t& stream,
                                                   const RotaryEmbeddingType rotary_embedding_type,
                                                   const float scaling_factor, const float low_freq_factor,
                                                   const float high_freq_factor, const int original_max_position_embeddings);
template void RotaryEmbeddingCuda<__hip_bfloat16>::SetConfig(
    __hip_bfloat16* cos_sin_cache, const int rotary_dim, const int max_position_embeddings, const float base,
    const int head_size, const int num_heads, const int num_kv_heads, const int stride_size, const bool is_neox,
    hipStream_t& stream, const RotaryEmbeddingType rotary_embedding_type, const float scaling_factor,
    const float low_freq_factor, const float high_freq_factor, const int original_max_position_embeddings);

}  // namespace nvidia
}  // namespace llm_kernels
