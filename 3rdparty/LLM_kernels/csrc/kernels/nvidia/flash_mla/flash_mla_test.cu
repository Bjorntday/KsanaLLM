/*
 * Copyright 2025 Tencent Inc.  All rights reserved.
 */

#include <random>
#include <sstream>

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include "csrc/kernels/nvidia/flash_mla/flash_mla.h"
#include "tests/kernels/nvidia/utils/testsuit_base.h"

namespace llm_kernels {
namespace nvidia {
namespace test {
class LlamaNvidiaFlashMlaTestSuit : public NvidiaTestSuitBase {
 public:
  void SetUp() override { NvidiaTestSuitBase::SetUp(); }

  void TearDown() override { NvidiaTestSuitBase::TearDown(); }

 protected:
  using NvidiaTestSuitBase::stream;
};

template <typename T>
inline void* CreateFlashMlaTensor(std::vector<int> shape) {
  size_t size = sizeof(T);
  for (int dim : shape) {
    size *= dim;
  }

  void* data_ptr;
  hipMalloc(&data_ptr, size);
  return data_ptr;
}

TEST_F(LlamaNvidiaFlashMlaTestSuit, FlashMlaKernelTest) {
  // 判断GPU是否是90以及以上的显卡
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);  // 获取设备0的属性

  int major = prop.major;
  int minor = prop.minor;

  std::cout << "当前GPU计算能力: " << major << "." << minor << std::endl;
  std::cout << "设备名称: " << prop.name << std::endl;

  if (major >= 9) {
    std::cout << "当前GPU是90或以上的显卡，支持Flash MLA操作" << std::endl;
  } else {
    std::cout << "当前GPU不是90或以上的显卡，可能不支持Flash MLA操作" << std::endl;
    GTEST_SKIP() << "跳过测试，因为当前GPU计算能力低于9.0";
  }

  int batch = 3;
  int num_heads = 16;
  int kv_lora_rank = 512;
  int qk_rope_head_dim = 64;
  int max_blocks_per_seq = 2;
  int block_num = 3;
  int num_kv_splits = 4;
  int page_size = 64;
  float sm_scale = 0.1147213867929261;

  void* q = CreateFlashMlaTensor<half>({batch, num_heads, kv_lora_rank + qk_rope_head_dim});
  void* k_buffer = CreateFlashMlaTensor<half>({block_num, page_size, 1, kv_lora_rank + qk_rope_head_dim});
  void* v_buffer = k_buffer;
  void* req_to_token = CreateFlashMlaTensor<int>({batch, max_blocks_per_seq});
  void* b_seqlen = CreateFlashMlaTensor<int>({batch});
  void* attn_out = CreateFlashMlaTensor<half>({batch, num_heads, num_kv_splits, kv_lora_rank + 1});

  // detail see ApplyWorkspaceBuffer
  void* workspace = CreateFlashMlaTensor<float>({4096});
  constexpr int q_seq_len = 1;
  InvokeFlashMla<half>(reinterpret_cast<half*>(q), reinterpret_cast<half*>(k_buffer), q_seq_len, sm_scale, req_to_token,
                       b_seqlen, nullptr, nullptr, workspace, attn_out, batch, num_heads, kv_lora_rank,
                       qk_rope_head_dim, page_size, max_blocks_per_seq, 0, block_num, stream);
}

}  // namespace test
}  // namespace nvidia
}  // namespace llm_kernels
