#include "hip/hip_runtime.h"

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include "csrc/kernels/nvidia/paged_attention/cache_copy.h"
namespace llm_kernels {
namespace nvidia {

#define MAX_THREADS_PER_BLOCK 1024

__device__ int k_chunk_size = 16;
/*
block_size：一个block块存几个token
block_offset：记录每个bs的block个数情况
total_len： 总的长度
*/
template <typename T>
__global__ void CacheCopyKernel(T* k_src, T* v_src, void** k_list, void** v_list, size_t* input_offst,
                                int* block_offset, int block_size, int bs, int total_len, int num_heads, int head_size,
                                int stride_size) {
  int x = k_chunk_size / sizeof(T);
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int hs_i = blockIdx.y;
  int head_size_i = hs_i / x;
  int j = hs_i % x;
  if (idx < total_len) {
    int block_idx = 0;
    for (block_idx = 0; block_idx < bs; block_idx++) {
      if (idx < input_offst[block_idx + 1]) {
        break;
      }
    }
    int cur_block_offset = (idx - input_offst[block_idx]) / block_size;
    int cur_batch_offset = (idx - input_offst[block_idx]) % block_size;
    T* k_dst_base = reinterpret_cast<T*>(k_list[block_offset[block_idx] + cur_block_offset]);
    T* v_dst_base = reinterpret_cast<T*>(v_list[block_offset[block_idx] + cur_block_offset]);
    T* k_src_ptr = k_src + idx * stride_size;
    T* v_src_ptr = v_src + idx * stride_size;

    for (int num_head_i = threadIdx.y; num_head_i < num_heads; num_head_i += blockDim.y) {
      int k_src_index = num_head_i * head_size + head_size_i * x + j;
      int k_dst_index =
          num_head_i * (head_size * block_size) + head_size_i * (block_size * x) + cur_batch_offset * x + j;
      int i = head_size_i * x + j;
      int v_src_index = num_head_i * head_size + i;
      int v_dst_index = num_head_i * (head_size * block_size) + i * block_size + cur_batch_offset;
      //  赋值操作
      k_dst_base[k_dst_index] = k_src_ptr[k_src_index];
      v_dst_base[v_dst_index] = v_src_ptr[v_src_index];
    }
  }
}

/*
block_size：一个block块存几个token
block_offset：记录每个bs的block个数情况
total_len： 总的长度
*/
template <typename T>
__global__ void CachePosCopyKernel(T* k_src, T* v_src, void** k_list, void** v_list, void* pos, size_t* input_offst,
                                   int* block_offset, int block_size, int bs, int total_len, int num_heads,
                                   int head_size, int stride_size) {
  int x = k_chunk_size / sizeof(T);
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int hs_i = blockIdx.y;
  int head_size_i = hs_i / x;
  int j = hs_i % x;
  if (idx < total_len) {
    int input_len = reinterpret_cast<int64_t*>(pos)[idx];
    int cur_block_offset = input_len / block_size;
    int cur_batch_offset = input_len % block_size;
    T* k_dst_base = reinterpret_cast<T*>(k_list[block_offset[idx] + cur_block_offset]);
    T* v_dst_base = reinterpret_cast<T*>(v_list[block_offset[idx] + cur_block_offset]);
    T* k_src_ptr = k_src + idx * stride_size;
    T* v_src_ptr = v_src + idx * stride_size;

    for (int num_head_i = threadIdx.y; num_head_i < num_heads; num_head_i += blockDim.y) {
      int k_src_index = num_head_i * head_size + head_size_i * x + j;
      int k_dst_index =
          num_head_i * (head_size * block_size) + head_size_i * (block_size * x) + cur_batch_offset * x + j;
      int i = head_size_i * x + j;
      int v_src_index = num_head_i * head_size + i;
      int v_dst_index = num_head_i * (head_size * block_size) + i * block_size + cur_batch_offset;
      //  赋值操作
      k_dst_base[k_dst_index] = k_src_ptr[k_src_index];
      v_dst_base[v_dst_index] = v_src_ptr[v_src_index];
    }
  }
}

template <typename T>
void CacheCopy(T* k_src, T* v_src, void** k_list, void** v_list, size_t* input_offst, int* block_offset, int block_size,
               int bs, int total_len, int num_heads, int head_size, int stride_size, hipStream_t stream) {
  int threadsPerBlock = 32;
  int min_num_heads = std::min(MAX_THREADS_PER_BLOCK / threadsPerBlock, num_heads);
  int blocks = (total_len + threadsPerBlock - 1) / threadsPerBlock;
  dim3 grid_shape(blocks, head_size);
  dim3 block_shape(threadsPerBlock, min_num_heads);
  CacheCopyKernel<<<grid_shape, block_shape, 0, stream>>>(k_src, v_src, k_list, v_list, input_offst, block_offset,
                                                          block_size, bs, total_len, num_heads, head_size, stride_size);
}

template <typename T>
void CachePosCopy(T* k_src, T* v_src, void** k_list, void** v_list, void* pos, size_t* input_offst, int* block_offset,
                  int block_size, int bs, int total_len, int num_heads, int head_size, int stride_size,
                  hipStream_t stream) {
  int threadsPerBlock = 32;
  int min_num_heads = std::min(MAX_THREADS_PER_BLOCK / threadsPerBlock, num_heads);
  int blocks = (total_len + threadsPerBlock - 1) / threadsPerBlock;
  dim3 grid_shape(blocks, head_size);
  dim3 block_shape(threadsPerBlock, min_num_heads);
  CachePosCopyKernel<<<grid_shape, block_shape, 0, stream>>>(k_src, v_src, k_list, v_list, pos, input_offst,
                                                             block_offset, block_size, bs, total_len, num_heads,
                                                             head_size, stride_size);
}

template void CacheCopy<float>(float* k_src, float* v_src, void** k_list, void** v_list, size_t* input_offst,
                               int* block_offset, int block_size, int bs, int total_len, int num_heads, int head_size,
                               int stride_size, hipStream_t stream);
template void CacheCopy<__hip_bfloat16>(__hip_bfloat16* k_src, __hip_bfloat16* v_src, void** k_list, void** v_list,
                                       size_t* input_offst, int* block_offset, int block_size, int bs, int total_len,
                                       int num_heads, int head_size, int stride_size, hipStream_t stream);
template void CacheCopy<half>(half* k_src, half* v_src, void** k_list, void** v_list, size_t* input_offst,
                              int* block_offset, int block_size, int bs, int total_len, int num_heads, int head_size,
                              int stride_size, hipStream_t stream);

template void CachePosCopy<float>(float* k_src, float* v_src, void** k_list, void** v_list, void* pos,
                                  size_t* input_offst, int* block_offset, int block_size, int bs, int total_len,
                                  int num_heads, int head_size, int stride_size, hipStream_t stream);
template void CachePosCopy<__hip_bfloat16>(__hip_bfloat16* k_src, __hip_bfloat16* v_src, void** k_list, void** v_list,
                                          void* pos, size_t* input_offst, int* block_offset, int block_size, int bs,
                                          int total_len, int num_heads, int head_size, int stride_size,
                                          hipStream_t stream);
template void CachePosCopy<half>(half* k_src, half* v_src, void** k_list, void** v_list, void* pos, size_t* input_offst,
                                 int* block_offset, int block_size, int bs, int total_len, int num_heads, int head_size,
                                 int stride_size, hipStream_t stream);

}  // namespace nvidia
}  // namespace llm_kernels
