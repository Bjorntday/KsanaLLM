#include "hip/hip_runtime.h"

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include "csrc/kernels/nvidia/paged_attention/cache_copy.h"
#include "csrc/kernels/nvidia/paged_attention/quant_utils.cuh"
namespace llm_kernels {
namespace nvidia {

#define MAX_THREADS_PER_BLOCK 1024

__device__ int k_chunk_size = 16;
/*
  block_size:     Number of tokens stored in each block.
  block_offsets:  Records the number of blocks for each batch size   [bs + 1,]
  prefix_offsets: Records the prefix length for each batch size (bs)
                  (accumulated from 0 to the current batch).         [bs + 1,]
*/
template <typename SCALAR_T, typename CACHE_T, bool FP8_E5M2>
__global__ void CacheCopyKernel(SCALAR_T* k_src, SCALAR_T* v_src, void** k_list, void** v_list, size_t* input_offsets,
                                size_t* prefix_offsets, int* block_offsets, int block_size, int bs, int total_len,
                                int num_heads, int head_size, int stride_size) {
  /*
    x:           In PagedAttention storage, KV-Blocks are divided into chunks to store head_size.
                 The variable x represents the size of each chunk.
    head_size_i: Indicates which chunk the head_size to be processed belongs to.
    j:           Represents the offset of the head_size to be processed within a single chunk.
  */
  int x = k_chunk_size / sizeof(SCALAR_T);
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int hs_i = blockIdx.y;
  int head_size_i = hs_i / x;
  int j = hs_i % x;
  if (idx < total_len) {
    int block_idx = 0;
    for (block_idx = 0; block_idx < bs; block_idx++) {
      if (idx < input_offsets[block_idx + 1]) {
        break;
      }
    }
    size_t prefix_limit = prefix_offsets[block_idx + 1] - prefix_offsets[block_idx] + input_offsets[block_idx];
    if (idx < prefix_limit) {
      return;
    }
    int cur_block_offset = (idx - input_offsets[block_idx]) / block_size;
    int cur_batch_offset = (idx - input_offsets[block_idx]) % block_size;
    CACHE_T* k_dst_base = reinterpret_cast<CACHE_T*>(k_list[block_offsets[block_idx] + cur_block_offset]);
    CACHE_T* v_dst_base = reinterpret_cast<CACHE_T*>(v_list[block_offsets[block_idx] + cur_block_offset]);
    SCALAR_T* k_src_ptr = k_src + idx * stride_size;
    SCALAR_T* v_src_ptr = v_src + idx * stride_size;

    for (int num_head_i = threadIdx.y; num_head_i < num_heads; num_head_i += blockDim.y) {
      int k_src_index = num_head_i * head_size + head_size_i * x + j;
      int k_dst_index =
          num_head_i * (head_size * block_size) + head_size_i * (block_size * x) + cur_batch_offset * x + j;
      int i = head_size_i * x + j;
      int v_src_index = num_head_i * head_size + i;
      int v_dst_index = num_head_i * (head_size * block_size) + i * block_size + cur_batch_offset;
      // Assignment operation
      if constexpr (FP8_E5M2) {
        k_dst_base[k_dst_index] = fp8_e5m2_unscaled::vec_conversion<CACHE_T, SCALAR_T>(k_src_ptr[k_src_index]);
        v_dst_base[v_dst_index] = fp8_e5m2_unscaled::vec_conversion<CACHE_T, SCALAR_T>(v_src_ptr[v_src_index]);
      } else {
        k_dst_base[k_dst_index] = k_src_ptr[k_src_index];
        v_dst_base[v_dst_index] = v_src_ptr[v_src_index];
      }
    }
  }
}

/*
  block_size:     Number of tokens stored in each block.
  block_offsets:  Records the number of blocks for each batch size   [bs + 1,]
  prefix_offsets: Records the prefix length for each batch size (bs)
                  (accumulated from 0 to the current batch).         [bs + 1,]
*/
template <typename SCALAR_T, typename CACHE_T, bool FP8_E5M2>
__global__ void ReverseCacheCopyKernel(SCALAR_T* k_src, SCALAR_T* v_src, void** k_list, void** v_list,
                                       size_t* input_offsets, size_t* prefix_offsets, int* block_offsets,
                                       int block_size, int bs, int total_len, int num_heads, int head_size,
                                       int stride_size) {
  /*
    x:           In PagedAttention storage, KV-Blocks are divided into chunks to store head_size.
                 The variable x represents the size of each chunk.
    head_size_i: Indicates which chunk the head_size to be processed belongs to.
    j:           Represents the offset of the head_size to be processed within a single chunk.
  */
  int x = k_chunk_size / sizeof(SCALAR_T);
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int hs_i = blockIdx.y;
  int head_size_i = hs_i / x;
  int j = hs_i % x;
  if (idx < total_len) {
    int block_idx = 0;
    for (block_idx = 0; block_idx < bs; block_idx++) {
      if (idx < input_offsets[block_idx + 1]) {
        break;
      }
    }
    size_t prefix_limit = prefix_offsets[block_idx + 1] - prefix_offsets[block_idx] + input_offsets[block_idx];
    if (idx >= prefix_limit) {
      return;
    }
    int cur_block_offset = (idx - input_offsets[block_idx]) / block_size;
    int cur_batch_offset = (idx - input_offsets[block_idx]) % block_size;
    CACHE_T* k_dst_base = reinterpret_cast<CACHE_T*>(k_list[block_offsets[block_idx] + cur_block_offset]);
    CACHE_T* v_dst_base = reinterpret_cast<CACHE_T*>(v_list[block_offsets[block_idx] + cur_block_offset]);
    SCALAR_T* k_src_ptr = k_src + idx * stride_size;
    SCALAR_T* v_src_ptr = v_src + idx * stride_size;

    for (int num_head_i = threadIdx.y; num_head_i < num_heads; num_head_i += blockDim.y) {
      int k_src_index = num_head_i * head_size + head_size_i * x + j;
      int k_dst_index =
          num_head_i * (head_size * block_size) + head_size_i * (block_size * x) + cur_batch_offset * x + j;
      int i = head_size_i * x + j;
      int v_src_index = num_head_i * head_size + i;
      int v_dst_index = num_head_i * (head_size * block_size) + i * block_size + cur_batch_offset;
      // Reverse assignment operation
      if constexpr (FP8_E5M2) {
        k_src_ptr[k_src_index] = fp8_e5m2_unscaled::vec_conversion<SCALAR_T, CACHE_T>(k_dst_base[k_dst_index]);
        v_src_ptr[v_src_index] = fp8_e5m2_unscaled::vec_conversion<SCALAR_T, CACHE_T>(v_dst_base[v_dst_index]);
      } else {
        k_src_ptr[k_src_index] = k_dst_base[k_dst_index];
        v_src_ptr[v_src_index] = v_dst_base[v_dst_index];
      }
    }
  }
}

/*
  block_size:    Number of tokens stored in each block.
  block_offsets: Records the number of blocks for each batch size   [bs + 1,]
*/
template <typename SCALAR_T, typename CACHE_T, bool FP8_E5M2>
__global__ void CachePosCopyKernel(SCALAR_T* k_src, SCALAR_T* v_src, void** k_list, void** v_list, void* pos,
                                   size_t* input_offsets, int* block_offsets, int block_size, int bs, int total_len,
                                   int num_heads, int head_size, int stride_size) {
  /*
    x:           In PagedAttention storage, KV-Blocks are divided into chunks to store head_size.
                 The variable x represents the size of each chunk.
    head_size_i: Indicates which chunk the head_size to be processed belongs to.
    j:           Represents the offset of the head_size to be processed within a single chunk.
  */
  int x = k_chunk_size / sizeof(SCALAR_T);
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int hs_i = blockIdx.y;
  int head_size_i = hs_i / x;
  int j = hs_i % x;
  if (idx < total_len) {
    int input_len = reinterpret_cast<int64_t*>(pos)[idx];
    int cur_block_offset = input_len / block_size;
    int cur_batch_offset = input_len % block_size;
    CACHE_T* k_dst_base = reinterpret_cast<CACHE_T*>(k_list[block_offsets[idx] + cur_block_offset]);
    CACHE_T* v_dst_base = reinterpret_cast<CACHE_T*>(v_list[block_offsets[idx] + cur_block_offset]);
    SCALAR_T* k_src_ptr = k_src + idx * stride_size;
    SCALAR_T* v_src_ptr = v_src + idx * stride_size;

    for (int num_head_i = threadIdx.y; num_head_i < num_heads; num_head_i += blockDim.y) {
      int k_src_index = num_head_i * head_size + head_size_i * x + j;
      int k_dst_index =
          num_head_i * (head_size * block_size) + head_size_i * (block_size * x) + cur_batch_offset * x + j;
      int i = head_size_i * x + j;
      int v_src_index = num_head_i * head_size + i;
      int v_dst_index = num_head_i * (head_size * block_size) + i * block_size + cur_batch_offset;
      //  赋值操作
      if constexpr (FP8_E5M2) {
        k_dst_base[k_dst_index] = fp8_e5m2_unscaled::vec_conversion<CACHE_T, SCALAR_T>(k_src_ptr[k_src_index]);
        v_dst_base[v_dst_index] = fp8_e5m2_unscaled::vec_conversion<CACHE_T, SCALAR_T>(v_src_ptr[v_src_index]);
      } else {
        k_dst_base[k_dst_index] = k_src_ptr[k_src_index];
        v_dst_base[v_dst_index] = v_src_ptr[v_src_index];
      }
    }
  }
}

template <typename SCALAR_T, typename CACHE_T, bool FP8_E5M2>
void CacheCopy(SCALAR_T* k_src, SCALAR_T* v_src, void** k_list, void** v_list, size_t* input_offsets,
               size_t* prefix_offsets, int* block_offsets, int block_size, int bs, int total_len, int num_heads,
               int head_size, int stride_size, hipStream_t stream) {
  int threadsPerBlock = 32;
  int min_num_heads = std::min(MAX_THREADS_PER_BLOCK / threadsPerBlock, num_heads);
  int blocks = (total_len + threadsPerBlock - 1) / threadsPerBlock;
  dim3 grid_shape(blocks, head_size);
  dim3 block_shape(threadsPerBlock, min_num_heads);
  CacheCopyKernel<SCALAR_T, CACHE_T, FP8_E5M2><<<grid_shape, block_shape, 0, stream>>>(
      k_src, v_src, k_list, v_list, input_offsets, prefix_offsets, block_offsets, block_size, bs, total_len, num_heads,
      head_size, stride_size);
}

template <typename SCALAR_T, typename CACHE_T, bool FP8_E5M2>
void ReverseCacheCopy(SCALAR_T* k_src, SCALAR_T* v_src, void** k_list, void** v_list, size_t* input_offsets,
                      size_t* prefix_offsets, int* block_offsets, int block_size, int bs, int total_len, int num_heads,
                      int head_size, int stride_size, hipStream_t stream) {
  int threadsPerBlock = 32;
  int min_num_heads = std::min(MAX_THREADS_PER_BLOCK / threadsPerBlock, num_heads);
  int blocks = (total_len + threadsPerBlock - 1) / threadsPerBlock;
  dim3 grid_shape(blocks, head_size);
  dim3 block_shape(threadsPerBlock, min_num_heads);
  ReverseCacheCopyKernel<SCALAR_T, CACHE_T, FP8_E5M2><<<grid_shape, block_shape, 0, stream>>>(
      k_src, v_src, k_list, v_list, input_offsets, prefix_offsets, block_offsets, block_size, bs, total_len, num_heads,
      head_size, stride_size);
}

template <typename SCALAR_T, typename CACHE_T, bool FP8_E5M2>
void CachePosCopy(SCALAR_T* k_src, SCALAR_T* v_src, void** k_list, void** v_list, void* pos, size_t* input_offsets,
                  int* block_offsets, int block_size, int bs, int total_len, int num_heads, int head_size,
                  int stride_size, hipStream_t stream) {
  int threadsPerBlock = 32;
  int min_num_heads = std::min(MAX_THREADS_PER_BLOCK / threadsPerBlock, num_heads);
  int blocks = (total_len + threadsPerBlock - 1) / threadsPerBlock;
  dim3 grid_shape(blocks, head_size);
  dim3 block_shape(threadsPerBlock, min_num_heads);
  CachePosCopyKernel<SCALAR_T, CACHE_T, FP8_E5M2>
      <<<grid_shape, block_shape, 0, stream>>>(k_src, v_src, k_list, v_list, pos, input_offsets, block_offsets,
                                               block_size, bs, total_len, num_heads, head_size, stride_size);
}

template <typename SCALAR_T, typename CACHE_T, bool FP8_E5M2>
__global__ void ConvertFP8AndBackKernel(SCALAR_T* data, size_t dim0, size_t dim1, int stride_size) {
  if constexpr (!FP8_E5M2) {
    return;
  }
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < dim0 * dim1) {
    // FP16 to FP8
    auto data_idx = idx / dim1 * stride_size + idx % dim1;
    CACHE_T temp = fp8_e5m2_unscaled::vec_conversion<CACHE_T, SCALAR_T>(data[data_idx]);
    // FP8 back to FP16
    data[data_idx] = fp8_e5m2_unscaled::vec_conversion<SCALAR_T, CACHE_T>(temp);
  }
}

template <typename SCALAR_T, typename CACHE_T, bool FP8_E5M2>
void ConvertFP8AndBack(SCALAR_T* data, size_t dim0, size_t dim1, int stride_size, hipStream_t stream) {
  int threads_per_block = 256;
  int blocks_per_grid = (dim0 * dim1 + threads_per_block - 1) / threads_per_block;
  ConvertFP8AndBackKernel<SCALAR_T, CACHE_T, FP8_E5M2>
      <<<blocks_per_grid, threads_per_block, 0, stream>>>(data, dim0, dim1, stride_size);
}

#define CACHE_COPY_FUNCTION_DECLARATION(SCALAR_T, CACHE_T, FP8_E5M2)                                                   \
  template void CacheCopy<SCALAR_T, CACHE_T, FP8_E5M2>(                                                                \
      SCALAR_T * k_src, SCALAR_T * v_src, void** k_list, void** v_list, size_t* input_offsets, size_t* prefix_offsets, \
      int* block_offsets, int block_size, int bs, int total_len, int num_heads, int head_size, int stride_size,        \
      hipStream_t stream);                                                                                            \
  template void ReverseCacheCopy<SCALAR_T, CACHE_T, FP8_E5M2>(                                                         \
      SCALAR_T * k_src, SCALAR_T * v_src, void** k_list, void** v_list, size_t* input_offsets, size_t* prefix_offsets, \
      int* block_offsets, int block_size, int bs, int total_len, int num_heads, int head_size, int stride_size,        \
      hipStream_t stream);                                                                                            \
  template void CachePosCopy<SCALAR_T, CACHE_T, FP8_E5M2>(                                                             \
      SCALAR_T * k_src, SCALAR_T * v_src, void** k_list, void** v_list, void* pos, size_t* input_offsets,              \
      int* block_offsets, int block_size, int bs, int total_len, int num_heads, int head_size, int stride_size,        \
      hipStream_t stream);                                                                                            \
  template void ConvertFP8AndBack<SCALAR_T, CACHE_T, FP8_E5M2>(SCALAR_T * data, size_t dim0, size_t dim1,              \
                                                               int stride_size, hipStream_t stream);

CACHE_COPY_FUNCTION_DECLARATION(float, float, false);
CACHE_COPY_FUNCTION_DECLARATION(float, uint8_t, true);
CACHE_COPY_FUNCTION_DECLARATION(half, half, false);
CACHE_COPY_FUNCTION_DECLARATION(half, uint8_t, true);
CACHE_COPY_FUNCTION_DECLARATION(__hip_bfloat16, __hip_bfloat16, false);
CACHE_COPY_FUNCTION_DECLARATION(__hip_bfloat16, uint8_t, true);
#undef CACHE_COPY_FUNCTION_DECLARATION

}  // namespace nvidia
}  // namespace llm_kernels
