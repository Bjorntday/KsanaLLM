#include "hip/hip_runtime.h"
/*
 * Copyright 2024 Tencent Inc.  All rights reserved.
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "permute.h"
#include "csrc/utils/nvidia/cuda_utils.h"

using namespace llm_kernels::utils;

namespace llm_kernels {
namespace nvidia {

template<size_t num_dims, typename IndexType>
PermuteKernelParams<num_dims, IndexType>
generatePermuteParams(const size_t* src_dims, const void* src, const size_t* permutation, void* dst, size_t count)
{
    PermuteKernelParams<num_dims, IndexType> params;
    params.src_index_helper = NdIndexOffsetHelper<IndexType, num_dims>(src_dims);
    size_t dst_dims[num_dims];
    for (size_t i = 0; i < num_dims; ++i) {
        dst_dims[i] = src_dims[permutation[i]];
    }
    params.dst_index_helper = NdIndexOffsetHelper<IndexType, num_dims>(dst_dims);
    for (size_t i = 0; i < num_dims; ++i) {
        params.permutation[i] = permutation[i];
    }
    params.src   = src;
    params.dst   = dst;
    params.count = static_cast<IndexType>(count);
    return params;
}

template<size_t num_dims, size_t movement_size, typename IndexType>
__global__ void PermuteKernel(PermuteKernelParams<num_dims, IndexType> params)
{
    using T       = typename std::aligned_storage<movement_size, movement_size>::type;
    const T*  src = reinterpret_cast<const T*>(params.src);
    T*        dst = reinterpret_cast<T*>(params.dst);
    IndexType src_index[num_dims];
    IndexType dst_index[num_dims];
    CUDA_1D_KERNEL_LOOP_T(IndexType, i, params.count)
    {
        params.dst_index_helper.OffsetToNdIndex(i, dst_index);
#pragma unroll
        for (size_t dim = 0; dim < num_dims; ++dim) {
            src_index[params.permutation[dim]] = dst_index[dim];
        }
        IndexType src_offset = params.src_index_helper.NdIndexToOffset(src_index);
        dst[i]               = src[src_offset];
    }
}

template <size_t num_dims, size_t movement_size>
void InvokePermute(void* input, void* output, std::vector<size_t> input_shape, std::vector<size_t> permutation,
                   hipStream_t& stream) {
    size_t total_size = 1;
    for (size_t& dim : input_shape) {
        total_size *= dim;
    }
    PermuteKernelParams<num_dims, size_t> permute_params = generatePermuteParams<num_dims, size_t>(
        const_cast<const size_t*>(input_shape.data()), const_cast<const void*>(input),
        const_cast<const size_t*>(permutation.data()), output, total_size);

    size_t kDefaultCudaThreadsNumPerBlock = 512;
    PermuteKernel<num_dims, movement_size, size_t>
        <<<BlocksNum4ThreadsNum(permute_params.count), kDefaultCudaThreadsNumPerBlock, 0, stream>>>(permute_params);
}

template
void InvokePermute<4ul, 4ul>(void*, void*, std::vector<size_t>, std::vector<size_t>, hipStream_t&);
template
void InvokePermute<3ul, 4ul>(void*, void*, std::vector<size_t>, std::vector<size_t>, hipStream_t&);
template
void InvokePermute<4ul, 2ul>(void*, void*, std::vector<size_t>, std::vector<size_t>, hipStream_t&);
template
void InvokePermute<3ul, 2ul>(void*, void*, std::vector<size_t>, std::vector<size_t>, hipStream_t&);

}  // namespace nvidia
}  // namespace llm_kernels
